#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <ctime>
#include <random>
#include <benchmark/benchmark.h>

#define THREADS_PER_BLOCK 512

static const int fromRange = 256;
static const int toRange = 1<<20;
// Kernel function to add the elements of two arrays
__global__
void dot(float *x, float *y, float* result)
{
  __shared__ float temp[THREADS_PER_BLOCK];
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  temp[threadIdx.x] = x[index] * y[index];
  __syncthreads();
  if( 0 == threadIdx.x ) {
      float sum = 0;
      for( int i = 0; i < THREADS_PER_BLOCK; i++ )
        sum += temp[i];
      atomicAdd( result , sum );
    }

}

static void BM_GPU(benchmark::State& state)
{
  const int N = state.range (0);
  float *x, *y, *result;

  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));
  hipMallocManaged(&result, sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
      x[i] = (float) rand();
      y[i] = (float) rand();
    }

  // Run kernel on 1M elements on the GPU
  for (auto _ : state)
  {
    dot<<< N/THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>(x, y,  result);
    benchmark::DoNotOptimize(result);
  }
  hipFree(x);
  hipFree(y);
  hipFree(result);
}
// Register the function as a benchmark
BENCHMARK(BM_GPU)->Range(fromRange, toRange);

BENCHMARK_MAIN ();


